#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <SFML/Graphics.hpp>

#define wdt 1024
#define hgt 1024
#define pix wdt*hgt

// TODO: Edge cases 
__global__
void updateSand(int *data_curr, int *data_old, unsigned char *pixels) {
    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;
    if(ROW >= hgt-1 || COL >= wdt-1 || ROW <= 0 || COL <= 0) return;
    int NUM = ROW*wdt + COL; 

    data_old[NUM] = data_curr[NUM]%4 + data_curr[NUM+1]/4 + data_curr[NUM-1]/4 + data_curr[NUM-wdt]/4 + data_curr[NUM+wdt]/4;
    if(data_old[NUM] > 0) {
        if(data_old[NUM] % 4 == 1) {
            pixels[4*NUM] = 255; pixels[4*NUM+1] = 0; pixels[4*NUM+2] = 0; }
        if(data_old[NUM] % 4 == 2) {
            pixels[4*NUM] = 0; pixels[4*NUM+1] = 255; pixels[4*NUM+2] = 0; }
        if(data_old[NUM] % 4 == 3) {
            pixels[4*NUM] = 0; pixels[4*NUM+1] = 0; pixels[4*NUM+2] = 255; }
        if(data_old[NUM] % 4 == 0) {
            pixels[4*NUM] = 255; pixels[4*NUM+1] = 255; pixels[4*NUM+2] = 255; }
    } else {
        for(int j=0; j<3; j++) pixels[4*NUM+j] = 0;
    }
}

int main() { 
    // boiler // create Window
    sf::RenderWindow window(sf::VideoMode(wdt, hgt), "Hello Sandkasten!");
    // boiler // needed to draw image
    sf::Texture texture;
    texture.create(wdt, hgt); 
    sf::Sprite sprite(texture);

    // code
    int *data_p, *data_q;
    unsigned char *pixels;

    hipMallocManaged(&data_p, pix*sizeof(int));
    hipMallocManaged(&data_q, pix*sizeof(int));
    hipMallocManaged(&pixels, 4*pix*sizeof(sf::Uint8));

    for(int i=0; i<pix; i++) pixels[4*i+3] = 255; //set alpha to max
    // set towers
    data_p[hgt/2*wdt + wdt/2] = 2000000;
    // boiler // run program
    while (window.isOpen()) {

        int xy = 32;
        dim3 thread_grid_in_block(xy, xy);
        dim3 grid_of_blocks((wdt + 32 - 1)/xy, (wdt + 32 - 1)/xy);

        updateSand<<<grid_of_blocks, thread_grid_in_block>>>(data_p, data_q, pixels);
        hipDeviceSynchronize();
        int *temp = data_p; data_p=data_q; data_q = temp;

        // boiler // show sprite
        texture.update(pixels);  
        window.clear();
        window.draw(sprite);
        window.display();

        sf::Event event;
        while (window.pollEvent(event)) {
            if (event.type == sf::Event::Closed || 
                event.key.code == sf::Keyboard::Escape) {
                
                hipFree(data_p);
                hipFree(data_q);
                hipFree(pixels);
                window.close();
            }
        }
    }
    return 0;
}   